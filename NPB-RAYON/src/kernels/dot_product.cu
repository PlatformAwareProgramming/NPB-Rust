// dot_product.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

extern "C" {

    __global__ void init_x_gpu(double* x, int n) {

        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

        if(thread_id < n) {
            x[thread_id] = 1.0;
        }
    }

    __global__ void init_conj_grad_gpu(double* x, double* q, double* z, double* r, double* p, int n) {

        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

        if(thread_id < n) {
            q[thread_id] = 0;
            z[thread_id] = 0;
            r[thread_id] = x[thread_id];
            p[thread_id] = r[thread_id];
        }
    }
// Kernel CUDA para multiplicar os vetores
    __global__ void dot_product_kernel(const double* x, const double* y, double* partial_sum, int n) {
        __shared__ double share_data[256]; // Cache compartilhado para redução
        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
        int local_id = threadIdx.x;

        share_data[local_id] = 0.0;

        if(thread_id >= n) { return; }

        share_data[threadIdx.x] = x[thread_id] * y[thread_id];

        __syncthreads();
        for(int i=blockDim.x/2; i>0; i>>=1){
            if(local_id<i){share_data[local_id]+=share_data[local_id+i];}
            __syncthreads();
        }

        if(local_id == 0) { 
            partial_sum[blockIdx.x] = share_data[0]; 
        }
    }

    __global__ void csr_matvec_kernel(
        const double* a,
        const int* colidx,
        const int* rowstr,
        const double* x,
        double* y,
        int num_rows
    ) {
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        if (row < num_rows) {
            double sum = 0.0;
            int start = rowstr[row];
            int end = rowstr[row + 1];
    
            for (int i = start; i < end; ++i) {
                sum += a[i] * x[colidx[i]];
            }
    
            y[row] = sum;
        }
    }

    __global__ void scalarvecmul1_gpu(double alpha, const double* x, double* y, int n) {
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        if(i < n) { 
            y[i] = x[i] + alpha*y[i];
        }
    }

    __global__ void scalarvecmul2_gpu(double alpha, const double* x, double* y, int n) {
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        if(i < n) { 
            y[i] += alpha*x[i];
        }
    }

    __global__ void norm_gpu(const double* x, const double* y, double* partial_sum, int n) {

        __shared__ double share_data[256]; // Cache compartilhado para redução
        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
        int local_id = threadIdx.x;

        share_data[local_id] = 0.0;

        if(thread_id >= n) { return; }

        { 
            double d;
            d = x[thread_id] - y[thread_id]; 
            share_data[threadIdx.x] = d * d;
        }

        __syncthreads();
        for(int i = blockDim.x/2; i>0; i>>=1) {
            if(local_id < i) { share_data[local_id] += share_data[local_id + i]; }
            __syncthreads();
        }

        if(local_id == 0) {
            partial_sum[blockIdx.x] = share_data[0]; 
        }
    }

    __global__ void update_x_gpu(double norm_temp2, const double* z, double* x, int n) {

        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

        if(thread_id < n) {
            x[thread_id] = norm_temp2 * z[thread_id];
        }
    }


int *d_colidx_;
int *d_rowstr_;
double *d_aa;
double *d_yy;
double *d_xx;
double *d_partial_sum;
double* h_partial_sum;

int *d_colidx;
void alloc_colidx_gpu(int** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_colidx = *x;
}

int *d_rowstr;
void alloc_rowstr_gpu(int** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_rowstr = *x;
}

double *d_a;
void alloc_a_gpu(double** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_a = *x;
}

double *d_x;
void alloc_x_gpu(double** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_x = *x;
}

double *d_z;
void alloc_z_gpu(double** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_z = *x;
}

double *d_p;
void alloc_p_gpu(double** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_p = *x;
}

double *d_q;
void alloc_q_gpu(double** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_q = *x;
}

double *d_r;
void alloc_r_gpu(double** x, int m) {
    CUDA_CHECK(hipMalloc((void**)x, m * sizeof(double)));
    d_r = *x;
}

void alloc_vectors_gpu(int m, int n) {

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente

    CUDA_CHECK(hipMalloc((void**)&d_xx, n * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_yy, n * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_partial_sum, gridSize * sizeof(double)));

    CUDA_CHECK(hipMalloc(&d_aa, m * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_colidx_, m * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_rowstr_, (n+1) * sizeof(int)));

    h_partial_sum = (double*) malloc(gridSize * sizeof(double));
}

void free_vectors_gpu() {

    CUDA_CHECK(hipFree(d_colidx_));
    CUDA_CHECK(hipFree(d_rowstr_));
    CUDA_CHECK(hipFree(d_aa));
    CUDA_CHECK(hipFree(d_xx));
    CUDA_CHECK(hipFree(d_yy));
    CUDA_CHECK(hipFree(d_partial_sum));
    
    free(h_partial_sum);
}

void launch_init_x_gpu(double* x, int n)
{
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente

    if (blockSize & (blockSize - 1)) {
        fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
        exit(EXIT_FAILURE);
    }
 
    init_x_gpu<<<gridSize, blockSize>>>(x, n);
  
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel

}


void launch_init_conj_grad_gpu(double* x, double* q, double* z, double* r, double* p, int n)
{
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente

    if (blockSize & (blockSize - 1)) {
        fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
        exit(EXIT_FAILURE);
    }
 
    init_conj_grad_gpu<<<gridSize, blockSize>>>(x, q, z, r, p, n);
  
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel

}

// Função wrapper para ser chamada do Rust
void dot_product_gpu(const double* x, 
                     const double* y, 
                     double* result, 
                     int n) {

    int blockSize = 256;
    int GridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente

    if (blockSize & (blockSize - 1)) {
        fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
        exit(EXIT_FAILURE);
    }
 
    CUDA_CHECK(hipMemcpy(d_xx, x, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_yy, y, n * sizeof(double), hipMemcpyHostToDevice));

    dot_product_kernel<<<GridSize, blockSize>>>(d_xx, d_yy, d_partial_sum, n);
  
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel

    CUDA_CHECK(hipMemcpy(h_partial_sum, d_partial_sum, GridSize * sizeof(double), hipMemcpyDeviceToHost));
 
    *result = 0.0;
    for (int i = 0; i < GridSize; i++) {
        *result += h_partial_sum[i];
    }
}

void move_a_to_device_gpu(const int* h_colidx, const int* h_rowstr, const double* h_a, int nnz, int num_rows) {
    CUDA_CHECK(hipMemcpy(d_aa, h_a, nnz * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_colidx_, h_colidx, nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rowstr_, h_rowstr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
}

void launch_csr_matvec_mul(
    const double* h_a,
    const int* h_colidx,
    const int* h_rowstr,
    const double* h_x,
    double* h_y,
    int nnz,
    int num_rows,
    int x_len
) {
    // Alocar memória na GPU

    // Transferências de memória: host -> device (somente leitura)
    CUDA_CHECK(hipMemcpy(d_aa, h_a, nnz * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_colidx_, h_colidx, nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rowstr_, h_rowstr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_xx, h_x, x_len * sizeof(double), hipMemcpyHostToDevice));

    // Configuração do kernel
    int blockSize = 256;
    int gridSize = (num_rows + blockSize - 1) / blockSize;

    csr_matvec_kernel<<<gridSize, blockSize>>>(
        d_aa, d_colidx_, d_rowstr_, d_xx, d_yy, num_rows
    );

    // Sincronizar GPU (garante conclusão)
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel

    // Transferência de resultado: device -> host
    CUDA_CHECK(hipMemcpy(h_y, d_yy, num_rows * sizeof(double), hipMemcpyDeviceToHost));

    // Liberar memória na GPU
 }

 void launch_scalarvecmul1_gpu(
    const double alpha, 
    const double* x, 
    double* y, 
    int n) {

        int blockSize = 256;
        int gridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente
    
        if (blockSize & (blockSize - 1)) {
            fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
            exit(EXIT_FAILURE);
        }
     
        CUDA_CHECK(hipMemcpy(d_xx, x, n * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_yy, y, n * sizeof(double), hipMemcpyHostToDevice));
    
        scalarvecmul1_gpu<<<gridSize, blockSize>>>(alpha, d_xx, d_yy, n);
      
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel
    
        CUDA_CHECK(hipMemcpy(y, d_yy, n * sizeof(double), hipMemcpyDeviceToHost));     
 }

 void launch_scalarvecmul2_gpu(
    const double alpha, 
    const double* x, 
    double* y, 
    int n) {

        int blockSize = 256;
        int gridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente
    
        if (blockSize & (blockSize - 1)) {
            fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
            exit(EXIT_FAILURE);
        }
     
        CUDA_CHECK(hipMemcpy(d_xx, x, n * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_yy, y, n * sizeof(double), hipMemcpyHostToDevice));
    
        scalarvecmul2_gpu<<<gridSize, blockSize>>>(alpha, d_xx, d_yy, n);
      
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel
    
        CUDA_CHECK(hipMemcpy(y, d_yy, n * sizeof(double), hipMemcpyDeviceToHost));        

 }

void launch_norm_gpu(const double* x, 
                     const double* y, 
                     double* result, 
                     int n) {

    int blockSize = 256;
    int GridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente

    if (blockSize & (blockSize - 1)) {
        fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
        exit(EXIT_FAILURE);
    }
 
    CUDA_CHECK(hipMemcpy(d_xx, x, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_yy, y, n * sizeof(double), hipMemcpyHostToDevice));

    norm_gpu<<<GridSize, blockSize>>>(d_xx, d_yy, d_partial_sum, n);
  
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel

    CUDA_CHECK(hipMemcpy(h_partial_sum, d_partial_sum, GridSize * sizeof(double), hipMemcpyDeviceToHost));
 
    *result = 0.0;
    for (int i = 0; i < GridSize; i++) {
        *result += h_partial_sum[i];
    }
}

void launch_update_x_gpu(double norm_temp2, const double* z, double* x, int n)
{
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // Ajusta o número de blocos dinamicamente

    if (blockSize & (blockSize - 1)) {
        fprintf(stderr, "Erro: o número de threads por bloco deve ser uma potência de 2.\n");
        exit(EXIT_FAILURE);
    }
 
    update_x_gpu<<<gridSize, blockSize>>>(norm_temp2, z, x, n);
  
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError()); // Verifica erros no lançamento do kernel
}

}
